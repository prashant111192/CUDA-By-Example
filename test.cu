#include "hip/hip_runtime.h"
#include <iostream>
#include <stdio.h>
#include "./common/book.h"

__global__ void add(int a, int b, int *c)
{
    *c = a + b;
}

int main(void)
{
    int count;
    hipDeviceProp_t prop;
    HANDLE_ERROR(hipGetDeviceCount(&count));
    HANDLE_ERROR(hipGetDeviceProperties(&prop, count - 1));
    printf(" --- General Information for device %d ---\n", count);
    printf("Name: %s\n", prop.name);
    printf("Compute capability: %d.%d\n", prop.major, prop.minor);
    printf("Clock rate: %d\n", prop.clockRate);
    printf("Device copy overlap: ");
    if (prop.deviceOverlap)
        printf("Enabled\n");
    else
        printf("Disabled\n");
    printf("Kernel execition timeout : ");
    if (prop.kernelExecTimeoutEnabled)
        printf("Enabled\n");
    else
        printf("Disabled\n");
    printf(" --- Memory Information for device %d ---\n", count);
    printf("Total global mem: %ld\n", prop.totalGlobalMem);
    printf("Total constant Mem: %ld\n", prop.totalConstMem);
    printf("Max mem pitch: %ld\n", prop.memPitch);
    printf("Texture Alignment: %ld\n", prop.textureAlignment);
    printf(" --- MP Information for device %d ---\n", count);
    printf("Multiprocessor count: %d\n", prop.multiProcessorCount);
    printf("Shared mem per mp: %ld\n", prop.sharedMemPerBlock);
    printf("Registers per mp: %d\n", prop.regsPerBlock);
    printf("Threads in warp: %d\n", prop.warpSize);
    printf("Max threads per block: %d\n", prop.maxThreadsPerBlock);
    printf("Max thread dimensions: (%d, %d, %d)\n", prop.maxThreadsDim[0], prop.maxThreadsDim[1], prop.maxThreadsDim[2]);
    printf("Max grid dimensions: (%d, %d, %d)\n", prop.maxGridSize[0], prop.maxGridSize[1], prop.maxGridSize[2]);
    printf("\n");
    // std::cout<< "name "<< prop.name<< std::endl;
    // std::cout<< "totalGloblaMem "<< prop.totalGlobalMem<< std::endl;
    // std::cout<< "max threads dim "<< prop.maxThreadsDim[0]<< prop.maxThreadsDim[1]<< prop.maxThreadsDim[2]<<std::endl;
    // std::cout<< "max threads per block "<< prop.maxThreadsPerBlock<< std::endl;
    // std::cout<< "compute mode "<< prop.computeMode<< std::endl;
    // std::cout<< "max Grid size "<< prop.maxGridSize[1]<< std::endl;
    // std::cout<<"count is: "<<count<<std::endl;
    int c;
    int *dev_c;
    HANDLE_ERROR(hipMalloc((void **)&dev_c, sizeof(int)));
    add<<<1, 1>>>(2, 7, dev_c);
    HANDLE_ERROR(hipMemcpy(&c, dev_c, sizeof(int), hipMemcpyDeviceToHost));
    printf("2 + 7 = %d\n", c);
    hipFree(dev_c);
    return 0;
}
