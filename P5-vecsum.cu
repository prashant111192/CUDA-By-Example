#include "hip/hip_runtime.h"
#include <stdio.h>
#include <iostream>


#include "common/book.h"

#define N	303*10024

__global__ void add( int *a, int *b, int *c ) {
	// int tid = blockIdx.x;
	long tid = threadIdx.x + blockIdx.x *blockDim.x;	// handle the data at this index
	while(tid < N)
    {
		c[tid] = a[tid] + b[tid];
        tid = tid + gridDim.x*blockDim.x;
    }
}

int main( void ) {
	int a[N], b[N], c[N];
	int *dev_a, *dev_b, *dev_c;

	// allocate the memory on the cpu
	HANDLE_ERROR( hipMalloc( (void**)&dev_a, N * sizeof(int) ) );
	HANDLE_ERROR( hipMalloc( (void**)&dev_b, N * sizeof(int) ) );
	HANDLE_ERROR( hipMalloc( (void**)&dev_c, N * sizeof(int) ) );

	for( long i = 0; i < N; i++ ) {
		a[i] = i;
		b[i] = i * i;
	}

	HANDLE_ERROR( hipMemcpy( dev_a, a, N * sizeof(int), hipMemcpyHostToDevice ) );
	HANDLE_ERROR( hipMemcpy( dev_b, b, N * sizeof(int), hipMemcpyHostToDevice ) );

	add<<<128,128>>>( dev_a, dev_b, dev_c );

	HANDLE_ERROR( hipMemcpy( c, dev_c, N * sizeof(int), hipMemcpyDeviceToHost ) );

	bool success = true;

	for (long i =0; i<N; i++)
	{
		if (a[i]+ b[i]!=c[i])
		{
			success = false;
		}
	}

	std::cout<<sizeof(c)/4<<std::endl;
	if (success)
	{
		std::cout<<"it worked!!\n";
	}
	else
	{
		std::cout<<"damn you!!\n";
	}


	// for( int i = 0; i < N; i++ ){
	// 	printf( "%d + %d = %d\n", a[i], b[i], c[i] );
	// }

	hipFree( dev_a );
	hipFree( dev_b );
	hipFree( dev_c );

	return 0;
}


// int main (void)
// {
//     int a[N], b[N], c[N];
//     int *dev_a, *dev_b, *dev_c;

//     HANDLE_ERROR(hipMalloc((void **)&dev_a, N * sizeoff(int)));
//     HANDLE_ERROR(hipMalloc((void **)&dev_b, N * sizeoff(int)));
//     HANDLE_ERROR(hipMalloc((void **)&dev_c, N * sizeoff(int)));

//    for (int i =0; i<N; i++)
//    {
//     a[i] = i;
//     b[i] = i*i;
//    } 

//    HANDLE_ERROR
// }