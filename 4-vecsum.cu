#include "hip/hip_runtime.h"
#include "common/book.h"

#define N	65700

__global__ void add( int *a, int *b, int *c ) {
	int tid = blockIdx.x;	// handle the data at this index
	if(tid < N)
		c[tid] = a[tid] + b[tid];
}

int main( void ) {
	int a[N], b[N], c[N];
	int *dev_a, *dev_b, *dev_c;

	// allocate the memory on the cpu
	HANDLE_ERROR( hipMalloc( (void**)&dev_a, N * sizeof(int) ) );
	HANDLE_ERROR( hipMalloc( (void**)&dev_b, N * sizeof(int) ) );
	HANDLE_ERROR( hipMalloc( (void**)&dev_c, N * sizeof(int) ) );

	for( int i = 0; i < N; i++ ) {
		a[i] = -i;
		b[i] = i * i;
	}

	HANDLE_ERROR( hipMemcpy( dev_a, a, N * sizeof(int), hipMemcpyHostToDevice ) );
	HANDLE_ERROR( hipMemcpy( dev_b, b, N * sizeof(int), hipMemcpyHostToDevice ) );

	add<<<N,1>>>( dev_a, dev_b, dev_c );

	HANDLE_ERROR( hipMemcpy( c, dev_c, N * sizeof(int), hipMemcpyDeviceToHost ) );


	for( int i = 0; i < N; i++ ){
		printf( "%d + %d = %d\n", a[i], b[i], c[i] );
	}

	hipFree( dev_a );
	hipFree( dev_b );
	hipFree( dev_c );

	return 0;
}
