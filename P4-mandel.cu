#include "hip/hip_runtime.h"

// To run use the following
// nvcc 4-mandel.cu -o run.out -lGL -lGLU -lglut




#include "common/book.h"
#include "common/cpu_bitmap.h"

#define DIM 1000


// __global__ void kernel(unsigned char *ptr)
// {
//     int x = blockIdx.x;
//     int y = blockIdx.y;

//     int offset = x+y*gridDim.x;

//     int juliaValue = julia(x,y);
//     ptr[offset*4+0] = 255*juliaValue;
//     ptr[offset*4+0] = 0;
//     ptr[offset*4+0] = 0;
//     ptr[offset*4+0] = 255;
// }

int main (void)
{
    CPUBitmap bitmap(DIM, DIM);

    unsigned char *dev_bitmap;

    HANDLE_ERROR(hipMalloc((void**)&dev_bitmap, bitmap.image_size()));

    dim3 grid(DIM,DIM);
    kernel <<<grid, 1>>> (dev_bitmap);

    HANDLE_ERROR(hipMemcpy (bitmap.get_ptr(), dev_bitmap, bitmap.image_size(), hipMemcpyDeviceToHost));
    bitmap.display_and_exit();

    hipFree(dev_bitmap);
}